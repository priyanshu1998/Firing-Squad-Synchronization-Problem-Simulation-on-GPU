#include <iostream>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <stdio.h>

constexpr int ids = 557432;
int trans[ids];

void trans_init() {
    trans[557312] = 1;
    trans[528384] = 3;
    trans[65536] = 4;
    trans[0] = 0;
    trans[8] = 0;
    trans[136] = 0;
    trans[557364] = 1;
    trans[529216] = 7;
    trans[78848] = 4;
    trans[212992] = 3;
    trans[262144] = 4;
    trans[557428] = 1;
    trans[530243] = 3;
    trans[95284] = 7;
    trans[475968] = 3;
    trans[275456] = 4;
    trans[557367] = 1;
    trans[529267] = 3;
    trans[79668] = 4;
    trans[226115] = 6;
    trans[472116] = 4;
    trans[213824] = 3;
    trans[529222] = 7;
    trans[78948] = 3;
    trans[214595] = 7;
    trans[287796] = 4;
    trans[410435] = 3;
    trans[275508] = 4;
    trans[530231] = 7;
    trans[95092] = 4;
    trans[472899] = 3;
    trans[226356] = 7;
    trans[475971] = 3;
    trans[213827] = 3;
    trans[95287] = 6;
    trans[476019] = 3;
    trans[276276] = 4;
    trans[213000] = 3;
    trans[262280] = 1;
    trans[557366] = 1;
    trans[529251] = 3;
    trans[79412] = 7;
    trans[222022] = 3;
    trans[406628] = 3;
    trans[275505] = 5;
    trans[213784] = 0;
    trans[274824] = 1;
    trans[79667] = 7;
    trans[226103] = 3;
    trans[471924] = 4;
    trans[210755] = 3;
    trans[472119] = 3;
    trans[213875] = 3;
    trans[472117] = 5;
    trans[275717] = 5;
    trans[217175] = 0;
    trans[329073] = 7;
    trans[22296] = 0;
    trans[356744] = 1;
    trans[78947] = 3;
    trans[214579] = 6;
    trans[287540] = 4;
    trans[406342] = 3;
    trans[210021] = 5;
    trans[214608] = 1;
    trans[288005] = 4;
    trans[413776] = 0;
    trans[328967] = 5;
    trans[20592] = 6;
    trans[329473] = 5;
    trans[28696] = 0;
    trans[459144] = 1;
    trans[557427] = 1;
    trans[530230] = 7;
    trans[95076] = 3;
    trans[472643] = 7;
    trans[222261] = 7;
    trans[410449] = 0;
    trans[275732] = 7;
    trans[217408] = 1;
    trans[332805] = 7;
    trans[82006] = 3;
    trans[263525] = 7;
    trans[22096] = 7;
    trans[353537] = 0;
    trans[413720] = 7;
    trans[275509] = 5;
    trans[213840] = 0;
    trans[275713] = 5;
    trans[217112] = 7;
    trans[328072] = 1;
    trans[95095] = 7;
    trans[472944] = 1;
    trans[227079] = 1;
    trans[487537] = 7;
    trans[460567] = 7;
    trans[29043] = 1;
    trans[464695] = 7;
    trans[460568] = 7;
    trans[29064] = 1;
    trans[557431] = 2;
    trans[530289] = 2;
    trans[96017] = 2;
    trans[487703] = 2;
    trans[463223] = 2;
    trans[71537] = 2;
    trans[96023] = 2;
    trans[487799] = 2;
    trans[464753] = 2;
    trans[96024] = 2;
    trans[487816] = 2;
    trans[95281] = 1;
    trans[475928] = 0;
    trans[557361] = 2;
    trans[529168] = 2;
    trans[78081] = 2;
    trans[200728] = 2;
    trans[65928] = 2;
    trans[472113] = 5;
    trans[78949] = 7;
    trans[288001] = 7;
    trans[226357] = 1;
    trans[475984] = 0;
    trans[475920] = 0;
    trans[274693] = 1;
    trans[200791] = 3;
    trans[66929] = 1;
    trans[200723] = 2;
    trans[65841] = 2;
    trans[4880] = 2;
    trans[406629] = 5;
    trans[413783] = 0;
    trans[79669] = 1;
    trans[226129] = 1;
    trans[472340] = 0;
    trans[332807] = 3;
    trans[82032] = 1;
    trans[263937] = 1;
    trans[529169] = 2;
    trans[78096] = 2;
    trans[200961] = 2;
    trans[69651] = 2;
    trans[4881] = 2;
    trans[69656] = 2;
    trans[213776] = 0;
    trans[200784] = 3;
    trans[66823] = 4;
    trans[413715] = 7;
    trans[327988] = 1;
    trans[4934] = 7;
    trans[528392] = 3;
    trans[65672] = 1;
    trans[529176] = 2;
    trans[78216] = 2;
    trans[210020] = 3;
    trans[222259] = 3;
    trans[410423] = 3;
    trans[275316] = 4;
    trans[217168] = 0;
    trans[95091] = 3;
    trans[472883] = 7;
    trans[226100] = 4;
    trans[471875] = 3;
    trans[209969] = 5;
    trans[66821] = 4;
    trans[20566] = 0;
    trans[329061] = 5;
    trans[217107] = 7;
    trans[4928] = 7;
    trans[78853] = 4;
    trans[213079] = 3;
    trans[263536] = 1;
    trans[22279] = 0;
    trans[356465] = 5;
    trans[22295] = 0;
    trans[356724] = 1;
    trans[464707] = 3;
    trans[209975] = 3;
    trans[226355] = 6;
    trans[475955] = 3;
    trans[275252] = 4;
    trans[209734] = 3;
    trans[328965] = 5;
    trans[95286] = 6;
    trans[476003] = 3;
    trans[276020] = 7;
    trans[222019] = 3;
    trans[406581] = 5;
    trans[213841] = 0;
    trans[82000] = 3;
    trans[263429] = 4;
    trans[20567] = 0;
    trans[329072] = 7;
    trans[79415] = 6;
    trans[222067] = 3;
    trans[407349] = 1;
    trans[226128] = 1;
    trans[472327] = 0;
    trans[217201] = 6;
    trans[329495] = 0;
    trans[464692] = 3;
    trans[95040] = 6;
    trans[472071] = 3;
    trans[213104] = 1;
    trans[263941] = 1;
    trans[28759] = 0;
    trans[460145] = 6;
    trans[79409] = 1;
    trans[221969] = 0;
    trans[405776] = 1;
    trans[200966] = 1;
    trans[69728] = 3;
    trans[67073] = 1;
    trans[24595] = 0;
    trans[393526] = 1;
    trans[4963] = 3;
    trans[24600] = 0;
    trans[393608] = 1;
    trans[200721] = 2;
    trans[65811] = 2;
    trans[4401] = 2;
    trans[70416] = 2;
    trans[406579] = 3;
    trans[213815] = 3;
    trans[407347] = 7;
    trans[226099] = 3;
    trans[471860] = 4;
    trans[209731] = 3;
    trans[20560] = 0;
    trans[407348] = 4;
    trans[213072] = 3;
    trans[263431] = 4;
    trans[329477] = 5;
    trans[475974] = 3;
    trans[275557] = 5;
    trans[288006] = 4;
    trans[413792] = 0;
    trans[329217] = 7;
    trans[28691] = 0;
    trans[4979] = 3;
    trans[459063] = 1;
    trans[472115] = 3;
    trans[213811] = 3;
    trans[406627] = 3;
    trans[406339] = 3;
    trans[209973] = 5;
    trans[226102] = 3;
    trans[471908] = 3;
    trans[210499] = 7;
    trans[275719] = 5;
    trans[410448] = 0;
    trans[472069] = 4;
    trans[213078] = 3;
    trans[353542] = 0;
    trans[471927] = 5;
    trans[210800] = 1;
    trans[227077] = 1;
    trans[487510] = 4;
    trans[460128] = 0;
    trans[22017] = 7;
    trans[352275] = 0;
    trans[4964] = 3;
    trans[79427] = 7;
    trans[222263] = 3;
    trans[410487] = 5;
    trans[276336] = 1;
    trans[227072] = 1;
    trans[487431] = 4;
    trans[1793] = 7;
    trans[458864] = 0;
    trans[472337] = 0;
    trans[70663] = 3;
    trans[217364] = 1;
    trans[332096] = 1;
    trans[70417] = 2;
    trans[69649] = 2;
    trans[410419] = 3;
    trans[209971] = 3;
    trans[210739] = 7;
    trans[471923] = 3;
    trans[356359] = 5;
    trans[275461] = 4;
    trans[22272] = 0;
    trans[472118] = 3;
    trans[213859] = 3;
    trans[287543] = 3;
    trans[406387] = 3;
    trans[28752] = 0;
    trans[460039] = 0;
    trans[472325] = 0;
    trans[217174] = 0;
    trans[329056] = 5;
    trans[210741] = 1;
    trans[222260] = 4;
    trans[410432] = 3;
    trans[275463] = 3;
    trans[95075] = 3;
    trans[472627] = 6;
    trans[222001] = 5;
    trans[406289] = 0;
    trans[209168] = 1;
    trans[200960] = 1;
    trans[65623] = 4;
    trans[1392] = 6;
    trans[22273] = 0;
    trans[356371] = 7;
    trans[69637] = 3;
    trans[4980] = 7;
    trans[79683] = 3;
    trans[475953] = 5;
    trans[275217] = 0;
    trans[69638] = 3;
    trans[65637] = 4;
    trans[1616] = 6;
    trans[25857] = 0;
    trans[95077] = 7;
    trans[472656] = 1;
    trans[222465] = 7;
    trans[413713] = 7;
    trans[327955] = 1;
    trans[4404] = 1;
    trans[70470] = 7;
    trans[78944] = 7;
    trans[395031] = 7;
    trans[214535] = 1;
    trans[286833] = 7;
    trans[395032] = 7;
    trans[464694] = 7;
    trans[222007] = 3;
    trans[210740] = 4;
    trans[327781] = 5;
    trans[413702] = 0;
    trans[288000] = 4;
    trans[222004] = 4;
    trans[213830] = 3;
    trans[352369] = 0;
    trans[22023] = 7;
    trans[263520] = 7;
    trans[287797] = 7;
    trans[210019] = 3;
    trans[353536] = 0;
    trans[410422] = 3;
    trans[275300] = 3;
    trans[471863] = 3;
    trans[209783] = 5;
    trans[487504] = 4;
    trans[460037] = 0;
    trans[213879] = 5;
    trans[1376] = 0;
    trans[458838] = 0;
    trans[487429] = 4;
    trans[275511] = 3;
    trans[471889] = 0;
    trans[28785] = 0;
    trans[81927] = 3;
    trans[1799] = 7;
    trans[262256] = 7;
    trans[70656] = 7;
    trans[226101] = 7;
    trans[210193] = 7;
    trans[460561] = 7;
    trans[463219] = 1;
    trans[28951] = 1;
    trans[71479] = 7;
    trans[209779] = 3;
    trans[209780] = 4;
    trans[356357] = 5;
    trans[66816] = 4;
    trans[20487] = 0;
    trans[327792] = 5;
    trans[209974] = 3;
    trans[275255] = 3;
    trans[460032] = 0;
    trans[406321] = 5;
    trans[209681] = 0;
    trans[87] = 0;
    trans[65541] = 4;
    trans[69632] = 3;
    trans[276019] = 6;
    trans[222003] = 3;
    trans[65616] = 4;
    trans[1287] = 0;
    trans[329479] = 5;
    trans[472884] = 4;
    trans[213809] = 5;
    trans[79414] = 6;
    trans[222051] = 3;
    trans[222032] = 0;
    trans[406785] = 5;
    trans[217105] = 7;
    trans[407093] = 1;
    trans[394609] = 6;
    trans[24663] = 0;
    trans[70464] = 7;
    trans[78854] = 4;
    trans[213093] = 3;
    trans[263685] = 1;
    trans[213088] = 3;
    trans[263760] = 1;
    trans[413809] = 6;
    trans[25863] = 0;
    trans[95046] = 6;
    trans[472163] = 3;
    trans[214581] = 1;
    trans[287568] = 0;
    trans[221968] = 0;
    trans[200800] = 3;
    trans[405765] = 1;
    trans[274694] = 1;
    trans[22289] = 0;
    trans[463220] = 1;
    trans[71491] = 3;
    trans[356631] = 1;
    trans[406327] = 3;
    trans[407091] = 6;
    trans[406324] = 4;
    trans[413696] = 0;
    trans[327685] = 5;
    trans[407092] = 7;
    trans[263430] = 4;
    trans[329221] = 7;
    trans[20576] = 0;
    trans[276277] = 1;
    trans[460289] = 6;
    trans[28768] = 0;
    trans[263942] = 1;
    trans[213814] = 3;
    trans[471859] = 3;
    trans[209719] = 3;
    trans[458832] = 0;
    trans[1286] = 0;
    trans[471861] = 5;
    trans[209745] = 0;
    trans[81925] = 3;
    trans[262230] = 4;
    trans[1381] = 0;
    trans[353543] = 0;
    trans[472164] = 3;
    trans[287795] = 3;
    trans[410421] = 5;
    trans[275281] = 0;
    trans[81920] = 3;
    trans[262151] = 4;
    trans[1798] = 7;
    trans[112] = 0;
    trans[329489] = 0;
    trans[71476] = 3;
    trans[24593] = 0;
    trans[393491] = 1;
    trans[70499] = 3;
    trans[4406] = 1;
    trans[327687] = 5;
    trans[20480] = 0;
    trans[329478] = 5;
    trans[287539] = 3;
    trans[406323] = 3;
    trans[209713] = 5;
    trans[1285] = 0;
    trans[275507] = 3;
    trans[80] = 0;
    trans[210483] = 6;
    trans[471907] = 3;
    trans[406352] = 0;
    trans[210177] = 5;
    trans[222005] = 5;
    trans[212997] = 3;
    trans[262231] = 4;
    trans[22278] = 0;
    trans[356448] = 5;
    trans[222068] = 4;
    trans[407363] = 3;
    trans[275280] = 0;
    trans[475957] = 5;
    trans[212998] = 3;
    trans[262245] = 4;
    trans[25862] = 0;
    trans[471909] = 5;
    trans[210512] = 1;
    trans[222469] = 4;
    trans[275552] = 5;
    trans[214533] = 1;
    trans[286806] = 4;
    trans[394592] = 0;
    trans[222262] = 3;
    trans[410469] = 5;
    trans[276048] = 1;
    trans[214528] = 1;
    trans[286727] = 4;
    trans[393328] = 0;
    trans[28689] = 0;
    trans[70515] = 3;
    trans[459027] = 1;
    trans[4407] = 1;
    trans[209716] = 4;
    trans[263424] = 4;
    trans[20486] = 0;
    trans[22016] = 7;
    trans[352263] = 0;
    trans[275251] = 3;
    trans[1280] = 0;
    trans[209717] = 5;
    trans[262224] = 4;
    trans[213813] = 5;
    trans[262149] = 4;
    trans[86] = 0;
    trans[471888] = 0;
    trans[210183] = 5;
    trans[472064] = 4;
    trans[212999] = 3;
    trans[1797] = 7;
    trans[28758] = 0;
    trans[410483] = 3;
    trans[276275] = 7;
    trans[28679] = 0;
    trans[1792] = 7;
    trans[352273] = 0;
    trans[70500] = 3;
    trans[209715] = 3;
    trans[406325] = 5;
    trans[209744] = 0;
    trans[287542] = 3;
    trans[406371] = 3;
    trans[210485] = 1;
    trans[406789] = 5;
    trans[275462] = 4;
    trans[24656] = 0;
    trans[394503] = 0;
    trans[25856] = 0;
    trans[413703] = 0;
    trans[471878] = 3;
    trans[406288] = 0;
    trans[209157] = 1;
    trans[274688] = 1;
    trans[200709] = 3;
    trans[275216] = 0;
    trans[200710] = 3;
    trans[210484] = 7;
    trans[222006] = 3;
    trans[28678] = 0;
    trans[458853] = 0;
    trans[263936] = 1;
    trans[287537] = 5;
    trans[65632] = 4;
    trans[1543] = 6;
    trans[24689] = 0;
    trans[28677] = 0;
    trans[458839] = 0;
    trans[475959] = 3;
    trans[275315] = 3;
    trans[458848] = 0;
    trans[356369] = 7;
    trans[70516] = 7;
    trans[395025] = 7;
    trans[71478] = 7;
    trans[329472] = 5;
    trans[327776] = 5;
    trans[275299] = 3;
    trans[210181] = 5;
    trans[471862] = 3;
    trans[209765] = 5;
    trans[329223] = 7;
    trans[286800] = 4;
    trans[394502] = 0;
    trans[213861] = 5;
    trans[472628] = 7;
    trans[406582] = 3;
    trans[286725] = 4;
    trans[393302] = 0;
    trans[210196] = 7;
    trans[332800] = 7;
    trans[209763] = 3;
    trans[209764] = 3;
    trans[352261] = 0;
    trans[275319] = 5;
    trans[487424] = 4;
    trans[458759] = 0;
    trans[460038] = 0;
    trans[406583] = 3;
    trans[28672] = 0;
    trans[356352] = 5;
    trans[275254] = 3;
    trans[394496] = 0;
    trans[209680] = 0;
    trans[200704] = 3;
    trans[406326] = 3;
    trans[458757] = 0;
    trans[275249] = 5;
    trans[65542] = 4;
    trans[96] = 0;
    trans[1541] = 6;
    trans[276021] = 1;
    trans[263686] = 1;
    trans[394753] = 6;
    trans[24672] = 0;
    trans[458758] = 0;
    trans[101] = 0;
    trans[1542] = 6;
    trans[327686] = 5;
    trans[329222] = 7;
    trans[209718] = 3;
    trans[393296] = 0;
    trans[22022] = 7;
    trans[352352] = 0;
    trans[407107] = 7;
    trans[222052] = 3;
    trans[458752] = 0;
    trans[7] = 0;
    trans[6] = 0;
    trans[287541] = 5;
    trans[262240] = 4;
    trans[1536] = 6;
    trans[24583] = 0;
    trans[5] = 0;
    trans[24662] = 0;
    trans[410467] = 3;
    trans[327680] = 5;
    trans[275253] = 5;
    trans[262150] = 4;
    trans[263680] = 1;
    trans[24582] = 0;
    trans[393317] = 0;
    trans[24581] = 0;
    trans[393303] = 0;
    trans[475958] = 3;
    trans[393312] = 0;
    trans[329216] = 7;
    trans[356358] = 5;
    trans[406388] = 4;
    trans[275301] = 5;
    trans[286720] = 4;
    trans[393223] = 0;
    trans[24576] = 0;
    trans[352256] = 0;
    trans[393221] = 0;
    trans[393222] = 0;
    trans[393216] = 0;
    trans[352262] = 0;
    trans[406372] = 3;
    trans[557320] = 2;
    trans[528520] = 2;
    trans[65544] = 4;
    trans[78856] = 7;
    trans[213128] = 1;
    trans[262152] = 4;
    trans[275464] = 5;
    trans[472344] = 0;
    trans[217480] = 1;
    trans[275736] = 7;
    trans[329496] = 0;
    trans[352280] = 0;
    trans[356376] = 7;
}

#define KNRM  "\x1B[90m"
#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define KYEL  "\x1B[33m"
#define KBLU  "\x1B[34m"
#define KMAG  "\x1B[35m"
#define KCYN  "\x1B[36m"
#define KWHT  "\x1B[37m"

using namespace std;

extern int trans[];
extern const int ids;

void dbug(const char* format){
    std::cout << format;
}

template<typename T, typename... Targs>
void dbug(const char* format, T value, Targs... Fargs){
    for ( ; *format != '\0'; format++ ) {
        if ( *format == '%' ) {
            std::cout << value;
            dbug(format+1, Fargs...); // recursive call
            return;
        }
        std::cout << *format;
    }
}

//__device__ int get_state_id(int far_left, int near_left, int current, int near_right, int far_right){
//    return ((far_left<<16) + (near_left<<12) + (current<<8) + (near_right<<4) + (far_right));
//}

__global__ void FSSP_init(int *d_ptr, int squad_size){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < squad_size+4) {
        if (id == 0 || id == 1 || id == squad_size + 2 || id == squad_size + 3) {
            d_ptr[id] = 8; // 0b1000
        }
        else if (id == 2) {
            d_ptr[id] = 1;
        }
        else {
            d_ptr[id] = 0;
        }
    }
}

__global__ void get_next_state(int *d_squad, int *d_temp, int *d_trans, int n){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id>=2 && id < n+2) {
        int state_id = ((d_squad[id-2]<<16) + (d_squad[id-1]<<12) + (d_squad[id]<<8) + (d_squad[id+1]<<4) + (d_squad[id+2]));
        d_temp[id] = d_trans[state_id];
    }
}

__global__ void print_the_state(int *d_squad, int *d_temp, int n, int y){
    int id = blockDim.x * blockIdx.x + threadIdx.x;


    if(id >=2 && id < n+2) {
        d_squad[id] = d_temp[id];

        if (d_squad[id] == 0)
            printf("\033[%d;%dH%s\u2612",  y + 1, id - 1, KNRM);
        else if (d_squad[id] == 1)
            printf("\033[%d;%dH%s\u2612",  y + 1, id - 1, KGRN);
        else if (d_squad[id] == 2)
            printf("\033[%d;%dH%s\u2612",  y + 1, id - 1, KRED);
        else if (d_squad[id] == 3)
            printf("\033[%d;%dH%s\u2612",  y + 1, id - 1, KYEL);
        else if (d_squad[id] == 4)
            printf("\033[%d;%dH%s\u2612",  y + 1, id - 1, KMAG);
        else if (d_squad[id] == 5)
            printf("\033[%d;%dH%s\u2612",  y + 1, id - 1, KBLU);
        else if (d_squad[id] == 6)
            printf("\033[%d;%dH%s\u2612",  y + 1, id - 1, KCYN);
        else if (d_squad[id] == 7)
            printf("\033[%d;%dH%s\u2612",  y + 1, id - 1, KWHT);
        else
            printf("");

    }
    __syncthreads();
}

int main() {
    trans_init();
    int n;
    cout<<"Enter the length of the firing squad: "<<endl;
    cin>>n;

    int *d_squad;
    int *d_trans;
    int *d_temp;

    if(hipMalloc(&d_squad,sizeof(int)* (n+4)) != hipSuccess){
        dbug("Allocation Error | var: % | size: %. \n", "d_squad", n+4);
    }

    if(hipMalloc(&d_trans, sizeof(int)* ids) != hipSuccess){
        dbug("Allocation Error | var: % | size: %. \n", "d_trans", ids);
    }

    if(hipMemcpy(d_trans, trans, sizeof(int)*ids, hipMemcpyHostToDevice)){
        dbug("Memcopy Error | % -> % \n", "trans", "d_trans");
    }


    FSSP_init<<<1,n+4>>>(d_squad, n);

    if(hipMalloc(&d_temp,sizeof(int)*(n+4))!= hipSuccess){
        dbug("Allocation Error | var: % | size: %. \n", "d_temp", n+4);
    }


    //Print initial state

    for(int i=2; i<n+2; i++){
        if(i == 2){
            printf("%s\033[%d;%dH\u2612",KGRN, 1, i-1 );
        }
        else{
            printf("%s\033[%d;%dH\u2612",KWHT, 1, i-1 );
        }
    }
    cout<<endl;

    for(int i=1; i<n; i++) {
        get_next_state<<<1, n + 4>>>(d_squad, d_temp, d_trans, n);
        hipDeviceSynchronize();

        print_the_state<<<1, n + 4>>>(d_squad, d_temp, n, i);
        hipDeviceSynchronize();

        printf("\n");
    }


    hipFree(d_squad);
    hipFree(d_temp);
    hipFree(d_trans);
    return 0;
}
